#include <hip/hip_runtime.h>

extern "C" __global__ void add(float *a, float *b, float *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}